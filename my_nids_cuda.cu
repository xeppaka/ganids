#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "my_nids_cuda.h"

__device__ int check_payload_part(int payload, int cur_state, int result_rule, int task_num, int *dfa, int *dfa_offset, int pitch_int, int *out_buffer) {
    unsigned c1, c2, c3, c4;
    c1 = payload & 0xFF;
    c2 = (payload >> 8) & 0xFF;
    c3 = (payload >> 16) & 0xFF;
    c4 = (payload >> 24) & 0xFF;

    cur_state = (dfa_offset + cur_state * pitch_int)[c1];

    if (cur_state == -1)
        return -1;

    if (cur_state == -2) {
        out_buffer[task_num] = result_rule;
        return -1;
    }

    if ((dfa_offset + cur_state * pitch_int)[256] > 0) {
        if (dfa_offset == dfa)
            out_buffer[task_num] = cur_state;
        else
            out_buffer[task_num] = result_rule;

        return -1;
    }

    cur_state = (dfa_offset + cur_state * pitch_int)[c2];

    if (cur_state == -1)
        return -1;

    if (cur_state == -2) {
        out_buffer[task_num] = result_rule;
        return -1;
    }

    if ((dfa_offset + cur_state * pitch_int)[256] > 0) {
        if (dfa_offset == dfa)
            out_buffer[task_num] = cur_state;
        else
            out_buffer[task_num] = result_rule;

        return -1;
    }

    cur_state = (dfa_offset + cur_state * pitch_int)[c3];

    if (cur_state == -1)
        return -1;

    if (cur_state == -2) {
        out_buffer[task_num] = result_rule;
        return -1;
    }

    if ((dfa_offset + cur_state * pitch_int)[256] > 0) {
        if (dfa_offset == dfa)
            out_buffer[task_num] = cur_state;
        else
            out_buffer[task_num] = result_rule;

        return -1;
    }

    cur_state = (dfa_offset + cur_state * pitch_int)[c4];

    if (cur_state == -1)
        return -1;

    if (cur_state == -2) {
        out_buffer[task_num] = result_rule;
        return -1;
    }

    if ((dfa_offset + cur_state * pitch_int)[256] > 0) {
        if (dfa_offset == dfa)
            out_buffer[task_num] = cur_state;
        else
            out_buffer[task_num] = result_rule;

        return -1;
    }

    return cur_state;
}

__global__ void analyze_payload_cuda_(int *dfa,
                                      int pitch,
                                      unsigned char *packet_buffer,
                                      int *task_buffer,
                                      int *out_buffer,
                                      int tasks)
{
    int task_num = blockIdx.x * blockDim.x + threadIdx.x;
    int pitch_int = pitch / 4;

    while (task_num < tasks) {
        int task_offset = task_num * 4;
        out_buffer[task_num] = -1;

        unsigned char* payload = packet_buffer + task_buffer[task_offset];
        unsigned payload_length = task_buffer[task_offset + 1];
        int *dfa_offset = dfa + pitch_int * task_buffer[task_offset + 2];
        unsigned result_rule = task_buffer[task_offset + 3];
        unsigned cur_state = 0;
        unsigned cur_idx = 0;
        int ret;

        while (cur_idx < payload_length) {
            int4 payload_part = *((int4*)(payload + cur_idx));
            if ((ret = check_payload_part(payload_part.x, cur_state, result_rule, task_num, dfa, dfa_offset, pitch_int, out_buffer)) < 0)
                break;
            cur_state = ret;

            if ((ret = check_payload_part(payload_part.y, cur_state, result_rule, task_num, dfa, dfa_offset, pitch_int, out_buffer)) < 0)
                break;
            cur_state = ret;

            if ((ret = check_payload_part(payload_part.z, cur_state, result_rule, task_num, dfa, dfa_offset, pitch_int, out_buffer)) < 0)
                break;
            cur_state = ret;

            if ((ret = check_payload_part(payload_part.w, cur_state, result_rule, task_num, dfa, dfa_offset, pitch_int, out_buffer)) < 0)
                break;
            cur_state = ret;

            cur_idx += 16;
        }

        task_num += gridDim.x * blockDim.x;
    }
}

//void analyze_payload_cuda(int *dfa,
//                          int pitch,
//                          unsigned char *packet_buffer,
//                          int *task_buffer,
//                          int *out_buffer,
//                          int tasks)
//{
//    analyze_payload_cuda_<<< 8, 256 >>>(dfa, pitch, packet_buffer, task_buffer, out_buffer, tasks);
//}

//__global__ void analyze_payload_cuda_(int pitch,
//                                      unsigned char *packet_buffer,
//                                      int *task_buffer,
//                                      int *out_buffer,
//                                      int tasks)
//{
//    int task_num = blockIdx.x * blockDim.x + threadIdx.x;
//    int pitch_int = pitch / sizeof(int);
//    __shared__ unsigned int payload_cache[16];

//    while (task_num < /*tasks*/1000) {
//        int task_offset = task_num * 4;
//        out_buffer[task_num] = -1;

//        unsigned char* payload = packet_buffer + task_buffer[task_offset];
//        int payload_length = task_buffer[task_offset + 1];
//        int dfa_offset = task_buffer[task_offset + 2];
//        int result_rule = task_buffer[task_offset + 3];
//        int cur_state = 0;
//        int cur_idx = 0;
//        int cache_index = 1024;

//        while (cur_idx < /*payload_length*/100) {
//            if (cache_index >= 64) {
//                for (int i = 0; i < 16 && (cur_idx + i*4 < payload_length); ++i) {
//                    payload_cache[i] = ((unsigned int*)(payload + cur_idx))[i];
//                }
//                cache_index = 0;
//            }

//            if (cur_state == -1)
//                break;

//            if (cur_state == -2) {
//                out_buffer[task_num] = result_rule;
//                break;
//            }

//            tex1Dfetch(dfa_texture, 0);
            //int t = task_buffer[cur_idx];

//            if (tex1Dfetch(dfa_texture, dfa_offset * pitch + 256) > 0) {
//                if (dfa_offset == 0)
//                    out_buffer[task_num] = cur_state;
//                else
//                    out_buffer[task_num] = result_rule;

//                break;
//            }

            //cur_state = tex1Dfetch(dfa_texture, dfa_offset * pitch + ((unsigned char *)payload_cache)[cache_index]);

//            cur_state = dfa_state[((unsigned char *)payload_cache)[cache_index]];
//            cur_state = dfa_state[payload[cur_idx]];

//            ++cur_idx;
//            ++cache_index;
//        }

//        out_buffer[task_num] = task_num;

//        task_num += gridDim.x * blockDim.x;
//    }
//}

void analyze_payload_cuda(int *dfa,
                          int pitch,
                          unsigned char *packet_buffer,
                          int *task_buffer,
                          int *out_buffer,
                          int tasks)
{
    analyze_payload_cuda_<<< 8, 32 >>>(dfa, pitch, packet_buffer, task_buffer, out_buffer, tasks);
}

//void bind_texture(int *device_memory, int size) throw(int) {
//    if (hipBindTexture(NULL, dfa_texture, device_memory, size) != hipSuccess)
//        throw 1;
//    dfa_texture.normalized = false;
////    dfa_texture.filterMode = hipFilterModePoint;
////    dfa_texture.addressMode = hipAddressModeClamp;
//}

//void unbind_texture() {
//    hipUnbindTexture(dfa_texture);
//}
