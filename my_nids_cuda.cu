#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "my_nids_cuda.h"

__global__ void analyze_payload_cuda_(int *dfa,
                                      int pitch,
                                      unsigned char *packet_buffer,
                                      int *task_buffer,
                                      int *out_buffer,
                                      int tasks)
{
    int task_num = blockIdx.x * blockDim.x + threadIdx.x;
    int pitch_int = pitch / 4;
    int task_offset = task_num * 4;

    while (task_num < tasks) {
        out_buffer[task_num] = -1;

        unsigned char* payload = packet_buffer + task_buffer[task_offset];
        int payload_length = task_buffer[task_offset + 1];
        int *dfa_offset = dfa + pitch_int * task_buffer[task_offset + 2];
        int result_rule = task_buffer[task_offset + 3];
        int cur_state = 0;
        int cur_idx = 0;

        while (true) {
            if (cur_state == -1)
                break;

            if (cur_state == -2) {
                out_buffer[task_num] = result_rule;
                break;
            }

            if ((dfa_offset + cur_state * pitch_int)[256] > 0) {
                if (dfa_offset == dfa)
                    out_buffer[task_num] = cur_state;
                else
                    out_buffer[task_num] = result_rule;

                break;
            }

            cur_state = (dfa_offset + cur_state * pitch_int)[payload[cur_idx]];
            ++cur_idx;

            if (cur_idx < payload_length)
                break;
        }

        __syncthreads();
        task_num += gridDim.x * blockDim.x;
    }
}

void analyze_payload_cuda(int *dfa,
                          int pitch,
                          unsigned char *packet_buffer,
                          int *task_buffer,
                          int *out_buffer,
                          int tasks)
{
    analyze_payload_cuda_<<< 4, 64 >>>(dfa, pitch, packet_buffer, task_buffer, out_buffer, tasks);
}
