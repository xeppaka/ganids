#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "my_nids_cuda.h"

__global__ void analyze_payload_cuda_(int *dfa,
                                      int pitch,
                                      unsigned char *packet_buffer,
                                      int *task_buffer,
                                      int *out_buffer,
                                      int tasks)
{
    int task_num = blockIdx.x * blockDim.x + threadIdx.x;
    int pitch_int = pitch / 4;
    __shared__ unsigned int payload_cache[16];

    while (task_num < tasks) {
        int task_offset = task_num * 4;
        out_buffer[task_num] = -1;

        unsigned char* payload = packet_buffer + task_buffer[task_offset];
        int payload_length = task_buffer[task_offset + 1];
        int *dfa_offset = dfa + pitch_int * task_buffer[task_offset + 2];
        int result_rule = task_buffer[task_offset + 3];
        int cur_state = 0;
        int cur_idx = 0;
        int cache_index = 0x00FFFFFF;

        while (cur_idx < payload_length) {
            if (cache_index >= 64) {
                for (int i = 0; i < 16 && (cur_idx + i*4 < payload_length); ++i) {
                    payload_cache[i] = ((unsigned int*)(payload + cur_idx))[i];
                }
                cache_index = 0;
            }
            __syncthreads();

            if (cur_state == -1)
                break;

            if (cur_state == -2) {
                out_buffer[task_num] = result_rule;
                break;
            }

            if (dfa_state[256] > 0) {
                if (dfa_offset == dfa)
                    out_buffer[task_num] = cur_state;
                else
                    out_buffer[task_num] = result_rule;

                break;
            }

            cur_state = dfa_state[((unsigned char *)payload_cache)[cache_index]];
//            cur_state = dfa_state[payload[cur_idx]];

            ++cur_idx;
            ++cache_index;
        }

        task_num += gridDim.x * blockDim.x;
    }
}

//void analyze_payload_cuda(int *dfa,
//                          int pitch,
//                          unsigned char *packet_buffer,
//                          int *task_buffer,
//                          int *out_buffer,
//                          int tasks)
//{
//    analyze_payload_cuda_<<< 8, 256 >>>(dfa, pitch, packet_buffer, task_buffer, out_buffer, tasks);
//}

//__global__ void analyze_payload_cuda_(int pitch,
//                                      unsigned char *packet_buffer,
//                                      int *task_buffer,
//                                      int *out_buffer,
//                                      int tasks)
//{
//    int task_num = blockIdx.x * blockDim.x + threadIdx.x;
//    int pitch_int = pitch / sizeof(int);
//    __shared__ unsigned int payload_cache[16];

//    while (task_num < /*tasks*/1000) {
//        int task_offset = task_num * 4;
//        out_buffer[task_num] = -1;

//        unsigned char* payload = packet_buffer + task_buffer[task_offset];
//        int payload_length = task_buffer[task_offset + 1];
//        int dfa_offset = task_buffer[task_offset + 2];
//        int result_rule = task_buffer[task_offset + 3];
//        int cur_state = 0;
//        int cur_idx = 0;
//        int cache_index = 1024;

//        while (cur_idx < /*payload_length*/100) {
//            if (cache_index >= 64) {
//                for (int i = 0; i < 16 && (cur_idx + i*4 < payload_length); ++i) {
//                    payload_cache[i] = ((unsigned int*)(payload + cur_idx))[i];
//                }
//                cache_index = 0;
//            }

//            if (cur_state == -1)
//                break;

//            if (cur_state == -2) {
//                out_buffer[task_num] = result_rule;
//                break;
//            }

//            tex1Dfetch(dfa_texture, 0);
            //int t = task_buffer[cur_idx];

//            if (tex1Dfetch(dfa_texture, dfa_offset * pitch + 256) > 0) {
//                if (dfa_offset == 0)
//                    out_buffer[task_num] = cur_state;
//                else
//                    out_buffer[task_num] = result_rule;

//                break;
//            }

            //cur_state = tex1Dfetch(dfa_texture, dfa_offset * pitch + ((unsigned char *)payload_cache)[cache_index]);

//            cur_state = dfa_state[((unsigned char *)payload_cache)[cache_index]];
//            cur_state = dfa_state[payload[cur_idx]];

//            ++cur_idx;
//            ++cache_index;
//        }

//        out_buffer[task_num] = task_num;

//        task_num += gridDim.x * blockDim.x;
//    }
//}

void analyze_payload_cuda(int *dfa,
                          int pitch,
                          unsigned char *packet_buffer,
                          int *task_buffer,
                          int *out_buffer,
                          int tasks)
{
    analyze_payload_cuda_<<< 8, 256 >>>(dfa, pitch, packet_buffer, task_buffer, out_buffer, tasks);
}

//void bind_texture(int *device_memory, int size) throw(int) {
//    if (hipBindTexture(NULL, dfa_texture, device_memory, size) != hipSuccess)
//        throw 1;
//    dfa_texture.normalized = false;
////    dfa_texture.filterMode = hipFilterModePoint;
////    dfa_texture.addressMode = hipAddressModeClamp;
//}

//void unbind_texture() {
//    hipUnbindTexture(dfa_texture);
//}
